
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 1000

__global__ void add(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N) c[idx] = a[idx] + b[idx];
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));
    for(int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * i;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    int M = 256;
    add<<<(N + M - 1) / M, M>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++) printf("%d + %d = %d\n", a[i], b[i], c[i]);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}