#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 1000

__global__ void add(double *a, double *b, double *c) {
    double idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N) c[idx] = a[idx] + b[idx];
}

int main() {
    double a[N], b[N], c[N];
    double *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(double));
    hipMalloc((void**)&dev_b, N * sizeof(double));
    hipMalloc((void**)&dev_c, N * sizeof(double));
    for(int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * i;
    }
    hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    int M = 128;
    add<<<(N + M - 1) / M, M>>>(a, b, c);
    hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++) printf("%lf + %lf = %lf\n", a[i], b[i], c[i]);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
